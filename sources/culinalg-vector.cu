/*
 * Implements vector
 */

#include<cassert>

#include<headers/culinalg-vector.hpp>
#include<headers/culinalg-exceptions.hpp>

#include<sources/culinalg-cuheader.cuh>

clg::Vector::alloc_irepr_throw_()
{
    // Set irepr_ to valid temp state
    irepr_->reset();
    
    // Try to allocate. We keep the local variables void here, unnecessary to do implicit conversion
    void* _h_data, _d_data;
    clg::wrapCudaError<clg::HostAllocationFailedException>(hipHostMalloc(&_h_data, dim_*sizeof(float)));
    clg::wrapCudaError<clg::DeviceAllocationFailedException>(hipMalloc(&_d_data, dim_*sizeof(float)));

    // Allocation passed
    irepr_->host_data = _h_data;
    irepr_->device_data = _d_data;
}

clg::Vector::delloc_irepr_throw_()
{
    // Ensure valid state
    void* _h_data = irepr_->host_data;
    void* _d_data = irepr_->device_data;
    irepr_->reset();

    // Attempt to free, throw if fail
    if(_h_data) clg::wrapCudaError<clg::HostDellocationFailedException>(hipHostFree(_h_data));
    if(_d_data) clg::wrapCudaError<clg::DeviceDellocationFailedException>(hipFree(_d_data));
}

clg::Vector::Vector(size_t n) : dim_(n)
{
    // Make new CuData
    irepr_ = new CuData();

    // Try allocating
    alloc_irepr_throw_();

    // Set to 0 on the CPU
    float* _h_floats = (float*)irepr_->host_data;
    for(size_t i = 0; i < dim_; ++i) _h_floats[i] = 0f;
}

clg::Vector::~Vector()
{
    // Free data
    if(irepr_->host_data)
        hipHostFree(irepr_->host_data);
    if(irepr_->device_data)
        hipFree(irepr_->device_data);

    // delte irepr_
    delete irepr_;
}

clg::Vector::Vector(const Vector& other) : dim_(other.dim_)
{
    // Make new CuData
    irepr_ = new CuData();

    // Reset CuData to make sure we have valid state
    irepr_->reset();

    // Copy data
    copyCuData(*irepr_, *(other.irepr_), dim_*sizeof(float));
}

clg::Vector::Vector(Vector&& other) : dim_(other.dim_)
{
    // Make new CuData
    irepr_ = new CuData();

    // Just move the data pointers in irepr_
    irepr_->move_from(*(other.irepr_));
    
    // Leave other in valid state, not pointing to same data.
    irepr_->reset();
}

clg::Vector::operator=(const Vector& other)
{
    // Check dimensionality
    if(dim_ != other.dim_) throw clg::DimensionalityMismatchException(dim_, other.dim_);

    // Attempt to delete data in this, maintain strong exception guarantee
    delloc_irepr_throw_();

    // Copy data
    copyCuData(*irepr_, *(other.irepr_), dim_*sizeof(float));
}

clg::Vector::operator=(Vector&& other)
{
    // Check dimensionality
    if(dim_ != other.dim_) throw clg::DimensionalityMismatchException(dim_, other.dim_);

    // Attempt to delete data in this, maintain strong exception guarantee
    delloc_irepr_throw_();

    // Just move the data pointers in irepr_
    irepr_->move_from(*(other.irepr_));
}

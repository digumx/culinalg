/**
 * Implements several functions declared in source/culinalg-cuheader.cuh
 */

#include<sources/culinalg-cuheader.cuh>

template<class E> inline void clg::wrapCudaError(const CudaError_t& err)
{
    if(err != hipSuccess)
        throw E("CUDA Error: " + std::string(hipGetErrorName(err)) + ": " +
                std::string(hipGetErrorString(err)));
}

void clg::copyCuObject(const CuObject& dst, const CuObject& src)
{
    CudaError_t err; 
    if(dst.irepr_->host_data_synced)
    {
        if(src.irepr_->host_data_synced)
            err = hipMemcpy(dst.irepr_->host_data, src.irepr_->host_data, hipMemcpyHostToHost); //TODO bench
        else
            err = hipMemcpy(dst.irepr_->host_data, src.irepr_->device_data, hipMemcpyDeviceToHost);
    }
    else
    {
        if(src.irepr_->host_data_synced)
            err = hipMemcpy(dst.irepr_->device_data, src.irepr_->host_data, hipMemcpyHostToDevice);        
        else
            err = hipMemcpy(dst.irepr_->device_data, src.irepr_->device_data, hipMemcpyDeviceToDevice);
    }

    // Check for error
    clg::wrapCudaError<clg::CopyFailedException>(err); 
}

/**
 * Implements several functions declared in source/culinalg-cuheader.cuh
 */

#include<sources/culinalg-cuheader.cuh>

template<class E> inline void clg::wrapCudaError(const CudaError_t& err)
{
    if(err != hipSuccess)
        throw E("CUDA Error: " + std::string(hipGetErrorName(err)) + ": " +
                std::string(hipGetErrorString(err)));
}

void clg::copyCuData(const CuData& dst, const CuData& src, size_t count)
{
    // Check that CuDatas point to valid data.
    if(!dst.host_data || !src.host_data || !dst.device_data || !src.host_data)
        throw clg::CopyFailedException("CuData is invalid or points to no data");

    // Perform copy
    CudaError_t err; 
    if(dst.host_data_synced)
    {
        if(src.host_data_synced)
            err = hipMemcpy(dst.host_data, src.host_data, count, hipMemcpyHostToHost); //TODO bench
        else
            err = hipMemcpy(dst.host_data, src.device_data, count, hipMemcpyDeviceToHost);
    }
    else
    {
        if(src.host_data_synced)
            err = hipMemcpy(dst.device_data, src.host_data, count, hipMemcpyHostToDevice);        
        else    //TODO benc the following
            err = hipMemcpy(dst.device_data, src.device_data, count, hipMemcpyDeviceToDevice);
    }

    // Check for error
    clg::wrapCudaError<clg::CopyFailedException>(err); 
}

void clg::CuData::reset()
{
    host_data = nullptr;
    device_data = nullptr;
    host_data_synced = true;
}

void clg::CuData::move_from(const CuData& src)
{
    host_data = other.host_data;
    device_data = other.device_data;
    host_data_synced = other.host_data_synced;
}

void clg::CuData::memsync_host(size_t size)
{
    // Early return
    if(host_data_synced) return;

    // Try copying
    clg::wrapCudaError<clg::CopyFailedException>(hipMemcpy(host_data, device_data, size,
                hipMemcpyDeviceToHost));

    // Set sync flags
    host_data_synced = true;
}

void clg::CuData::memsync_device(size_t size)
{
    // Early return
    if(!host_data_synced) return;

    // Try copying
    clg::wrapCudaError<clg::CopyFailedException>(hipMemcpy(device_data, host_data, size,
                hipMemcpyHostToDevice));

    // Set sync flags
    host_data_synced = false;
}

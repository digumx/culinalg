/**
 * Implements several functions declared in source/culinalg-cuheader.cuh
 */

#include<sources/culinalg-cuheader.cuh>

template<class E> inline void clg::wrapCudaError(const CudaError_t& err)
{
    if(err != hipSuccess)
        throw E("CUDA Error: " + std::string(hipGetErrorName(err)) + ": " +
                std::string(hipGetErrorString(err)));
}

void clg::copyCuData(const CuData& dst, const CuData& src, size_t count)
{
    // Check that CuDatas point to valid data.
    if(!dst.host_data || !src.host_data || !dst.device_data || !src.host_data)
        throw clg::CopyFailedException("CuData is invalid or points to no data");

    // Perform copy
    CudaError_t err; 
    if(dst.host_data_synced)
    {
        if(src.host_data_synced)
            err = hipMemcpy(dst.host_data, src.host_data, count, hipMemcpyHostToHost); //TODO bench
        else
            err = hipMemcpy(dst.host_data, src.device_data, count, hipMemcpyDeviceToHost);
    }
    else
    {
        if(src.host_data_synced)
            err = hipMemcpy(dst.device_data, src.host_data, count, hipMemcpyHostToDevice);        
        else    //TODO benc the following
            err = hipMemcpy(dst.device_data, src.device_data, count, hipMemcpyDeviceToDevice);
    }

    // Check for error
    clg::wrapCudaError<clg::CopyFailedException>(err); 
}
